#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>

#include <hip/hip_runtime.h>

#include "../dietgpu/ans/GpuANSCodec.h"
#include "../dietgpu/utils/StackDeviceMemory.h"

#include "cudaCheck.h"

int n_print = 1;

void print() {
    printf("%d\n", n_print);
    n_print++;
}

int compress(const void* in, const uint32_t* insize, void* out, uint32_t* outsize) {
    //dietgpu::StackDeviceMemory res;
    // Compression configuration
    auto config = dietgpu::ANSCodecConfig();

    // Number of separate, independent compression problems
    uint32_t numInBatch;

    // Host array with addresses of device pointers comprising the input batch
    // to compress
    void* in_dgpu[1];
    // Host array with sizes of batch members
    uint32_t inSize[1];

    // Optional (can be null): region in device memory of size 256 words
    // containing pre-calculated symbol counts (histogram) of the data to be
    // compressed
    uint32_t* histogram_dev;

    // Host array with addresses of device pointers for the compressed output
    // arrays. Each out[i] must be a region of memory of size at least
    // getMaxCompressedSize(inSize[i])
    void* out_dgpu[1];
    // Device memory array of size numInBatch (optional)
    // Provides the size of actual used memory in each output compressed batch
    uint32_t* outSize_dev;

    // stream on the current device on which this runs
    hipStream_t stream;

    uint32_t maxsize = dietgpu::getMaxCompressedSize(*insize);
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDA_CHECK(hipMalloc(in_dgpu, *insize));
    CUDA_CHECK(hipMalloc(out_dgpu, maxsize));
    CUDA_CHECK(hipMalloc(&outSize_dev, sizeof(uint32_t)));
    CUDA_CHECK(hipMemcpy(in_dgpu, in, *insize, hipMemcpyHostToDevice));

    int device = 0;
    size_t allocPerDevice = maxsize;
    auto res = dietgpu::StackDeviceMemory(device, allocPerDevice);
    numInBatch = 1;
    *inSize = *insize;
    histogram_dev = nullptr;

    int t = clock();
    dietgpu::ansEncodeBatchPointer(
            res,
            config,
            numInBatch,
            (const void**)in_dgpu,
            inSize,
            histogram_dev,
            out_dgpu,
            outSize_dev,
            stream);
    t = clock() - t;

    CUDA_CHECK(hipMemcpy(outsize, outSize_dev, sizeof(uint32_t), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(out, *out_dgpu, *outsize, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(*in_dgpu));
    CUDA_CHECK(hipFree(*out_dgpu));
    CUDA_CHECK(hipFree(outSize_dev));

    return t;
}

int main(int argc, char* argv[]) {
    char* ifname = NULL;
    char* ofname = NULL;
    FILE* fp;
    void* orig;
    uint32_t origsize;
    void* comp;
    uint32_t compsize;
    uint32_t maxsize;
    int t;

    switch (argc) {
        case 2:
            ifname = argv[1];
            break;
        case 3:
            ifname = argv[1];
            ofname = argv[2];
            break;
        default:
            printf("Usage: %s INFILE [OUTFILE]\n", argv[0]);
            return 1;
    }

    fp = fopen(ifname, "rb");
    fseek(fp, 0L, SEEK_END);
    origsize = ftell(fp);
    maxsize = dietgpu::getMaxCompressedSize(origsize);
    rewind(fp);
    orig = malloc(origsize);
    comp = malloc(maxsize);
    fread(orig, origsize, 1, fp);
    fclose(fp);

    t = compress(orig, &origsize, comp, &compsize);

    /* t = clock();
    compress(orig, &origsize, comp, &compsize);
    t = clock() - t; */

    if (ofname) {
        fp = fopen(ofname, "wb");
        fwrite(comp, compsize, 1, fp);
        fclose(fp);
        printf("wrote %s\n", ofname);
    }

    printf("insize: %u\n", origsize);
    printf("outsize: %u\n", compsize);
    printf("ratio: %f\n", (float)compsize/origsize);
    printf("throughput (comp): %f MB/s\n", (float)origsize/t * CLOCKS_PER_SEC/1000000);

    free(orig);
    free(comp);
}
